//
// Created by mrhax on 9/23/24.
//

#include "Blockchain.cuh"
#include "Block.cuh"
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdint>
#include "../utils/constants.hpp"
#include <sstream>
#include "../utils/sha256.cuh"

// Blockchain constructor
Blockchain::Blockchain(int difficulty) : difficulty(difficulty) {
    const std::string s = "Genesis Block";
    char dataArr[MAX_DATA_SIZE] = {};
    memcpy(dataArr, s.c_str(), s.length());

    Block genesis_block(blockchain.size(), time(nullptr), dataArr);
    genesis_block.currentHash[0] = '0';
    blockchain.push_back(genesis_block);
}

// Add block to the blockchain
void Blockchain::addBlock( const std::string& data) {
    //TODO put in a for loop to iter for more hashes
    // and increase in steps of blockIdx.x * blockDim.x + threadIdx.x (=MINING_TOTAL_THREADS)

    std::cout << "addBlock called" << std::endl;

    // Reset the stop_flag before launching the next kernel
    int reset_value = 0;
    hipMemcpyToSymbol(HIP_SYMBOL(stop_flag), &reset_value, sizeof(int));

    char dataArr[MAX_DATA_SIZE] = {};
    memcpy(dataArr, data.c_str(), data.length());

    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    ///Block
    Block h_block(blockchain.size(), time(nullptr), dataArr);
    memcpy(h_block.previousBlockHash, blockchain.back().currentHash, 64); //TODO is already null terminated?

    std::stringstream ss;
    ss << h_block.blockIndex  << "\n" << h_block.timeOfCreation<< "\n" << h_block.previousBlockHash<< "\n" << dataArr << "\n";
    const std::string resulting = ss.str();

    std::cout << "Resulting block: " << std::endl;
    std::cout << resulting << std::endl;

    char* d_block_data;
    hipMalloc(&d_block_data, MAX_DATA_SIZE);
    hipMemcpy(d_block_data, resulting.c_str(), resulting.length(), hipMemcpyHostToDevice);
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    ///Output
    char h_output[65] = {}; //TODO change to a char[65] to contain the sha-256 hash

    char* d_output;
    hipMalloc(&d_output, sizeof(char) * 65);
    hipMemset(d_output, 0, sizeof(char) * 65);
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

    std::cout << "Calling kernel..." << std::endl;
    hashKernel<<<MINING_SM_BLOCKS, MINING_BLOCK_THREADS>>>(d_block_data, MINING_TOTAL_THREADS, resulting.length(), d_output);
    hipDeviceSynchronize();

    hipMemcpy(h_output, d_output, sizeof(char)*65, hipMemcpyDeviceToHost);

    std::cout << "Resulting hash" << std::endl;
    std::cout << h_output << std::endl;

    hipFree(d_output);
    blockchain.push_back(h_block);
}
